#include <hipblas.h>
#include <hipsolver.h>
#include <iostream>
#include "cuutils.h"
#include "absnf.h"
#include "utils.hpp"
#include <chrono>
#include <typeinfo>
#define t_def double

typedef std::chrono::high_resolution_clock::time_point TimeVar;

void single_execution(int s)
{
	t_def *h_a = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_b = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_Z = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_L = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_J = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_Y = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_dx = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_dz = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_dy = (t_def *)malloc(s*sizeof(t_def));

	t_def *d_a; hipMalloc((void **)&d_a, s*sizeof(t_def));
	t_def *d_b; hipMalloc((void **)&d_b, s*sizeof(t_def));
	t_def *d_Z; hipMalloc((void **)&d_Z, s*s*sizeof(t_def));
	t_def *d_L; hipMalloc((void **)&d_L, s*s*sizeof(t_def));
	t_def *d_J; hipMalloc((void **)&d_J, s*s*sizeof(t_def));
	t_def *d_Y; hipMalloc((void **)&d_Y, s*s*sizeof(t_def));		
	t_def *d_dx; hipMalloc((void **)&d_dx, s*sizeof(t_def));
	t_def *d_dz; hipMalloc((void **)&d_dz, s*sizeof(t_def));
	t_def *d_abs_dz; hipMalloc((void **)&d_abs_dz, s*sizeof(t_def));
	t_def *d_dy; hipMalloc((void **)&d_dy, s*sizeof(t_def));

	utils::fillRandVector(h_a, s,-10,10);
	utils::fillRandVector(h_b, s,-10,10);
	utils::fillRandVector(h_Z, s*s,-10,10);
	utils::fillRandVector(h_J, s*s,-10,10);
	utils::fillRandVector(h_Y, s*s,-10,10);
	utils::fillRandVector(h_dx, s,-10,10);
	utils::fillRandMatrix(h_L, s,s,-10,10,0,utils::MATRIXOPT::LOWER);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	TimeVar t_0 = std::chrono::high_resolution_clock::now();

	hipMemcpy(d_a, h_a,  s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b,  s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_Z, h_Z,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_L, h_L,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_J, h_J,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, h_Y,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_dx, h_dx, s*sizeof(t_def), hipMemcpyHostToDevice);

	TimeVar t_1 = std::chrono::high_resolution_clock::now();
	absnf::eval_core(cublas_handle, d_a, d_b,
			 		  	 d_Z, d_L,
			 		  	 d_J, d_Y,
			 		  	 d_dx,
			 		  	 s, s, s,
			 		  	 d_dz, d_dy,
			 		  	 d_abs_dz);
	hipDeviceSynchronize();
	TimeVar t_2 = std::chrono::high_resolution_clock::now();
	
	hipMemcpy(h_dz, d_dz, s*sizeof(t_def), hipMemcpyDeviceToHost);
	hipMemcpy(h_dy, d_dy, s*sizeof(t_def), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	TimeVar t_3 = std::chrono::high_resolution_clock::now();
	
	auto int_upload = std::chrono::duration_cast<std::chrono::milliseconds>( t_1 - t_0 ).count();
	auto int_exec = std::chrono::duration_cast<std::chrono::milliseconds>( t_2 - t_1 ).count();
	auto int_download = std::chrono::duration_cast<std::chrono::milliseconds>( t_3 - t_2 ).count();
	auto int_total = std::chrono::duration_cast<std::chrono::milliseconds>( t_3 - t_0 ).count();
	
	std::cout << "----" << s << "----" << std::endl;
	std::cout <<"upload:  " << int_upload << std::endl;
	std::cout <<"exec:  " << int_exec << std::endl;
	std::cout <<"download:  " << int_download << std::endl;
	std::cout <<"total:  " << int_total << std::endl;
	free(h_a);
	free(h_b);
	free(h_Z);
	free(h_L);
	free(h_J);
	free(h_Y);
	free(h_dx);

	hipFree(d_a); 
	hipFree(d_b);
	hipFree(d_Z);
	hipFree(d_L);
	hipFree(d_J);
	hipFree(d_Y);
	hipFree(d_dx);
	hipFree(d_dz);
	hipFree(d_abs_dz);
	hipFree(d_dy);

	hipblasDestroy(cublas_handle);		
}

void multiple_executions(int s, int executions)
{
	t_def *h_a = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_b = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_Z = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_L = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_J = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_Y = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_dx = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_dz = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_dy = (t_def *)malloc(s*sizeof(t_def));

	t_def *d_a; hipMalloc((void **)&d_a, s*sizeof(t_def));
	t_def *d_b; hipMalloc((void **)&d_b, s*sizeof(t_def));
	t_def *d_Z; hipMalloc((void **)&d_Z, s*s*sizeof(t_def));
	t_def *d_L; hipMalloc((void **)&d_L, s*s*sizeof(t_def));
	t_def *d_J; hipMalloc((void **)&d_J, s*s*sizeof(t_def));
	t_def *d_Y; hipMalloc((void **)&d_Y, s*s*sizeof(t_def));		
	t_def *d_dx; hipMalloc((void **)&d_dx, s*sizeof(t_def));
	t_def *d_dz; hipMalloc((void **)&d_dz, s*sizeof(t_def));
	t_def *d_abs_dz; hipMalloc((void **)&d_abs_dz, s*sizeof(t_def));
	t_def *d_dy; hipMalloc((void **)&d_dy, s*sizeof(t_def));

	utils::fillRandVector(h_a, s,-10,10);
	utils::fillRandVector(h_b, s,-10,10);
	utils::fillRandVector(h_Z, s*s,-10,10);
	utils::fillRandVector(h_J, s*s,-10,10);
	utils::fillRandVector(h_Y, s*s,-10,10);
	utils::fillRandVector(h_dx, s,-10,10);
	utils::fillRandMatrix(h_L, s,s,-10,10,0,utils::MATRIXOPT::LOWER);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	// TimeVar t_0 = std::chrono::high_resolution_clock::now();

	hipMemcpy(d_a, h_a,  s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b,  s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_Z, h_Z,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_L, h_L,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_J, h_J,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, h_Y,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_dx, h_dx, s*sizeof(t_def), hipMemcpyHostToDevice);

	TimeVar t_0 = std::chrono::high_resolution_clock::now();
	for(int i=0; i<executions; i++)
	{
		absnf::eval_core(cublas_handle, d_a, d_b,
			 		  	 d_Z, d_L,
			 		  	 d_J, d_Y,
			 		  	 d_dx,
			 		  	 s, s, s,
			 		  	 d_dz, d_dy,
			 		  	 d_abs_dz);
	}
	hipDeviceSynchronize();
	TimeVar t_1 = std::chrono::high_resolution_clock::now();
	
	hipMemcpy(h_dz, d_dz, s*sizeof(t_def), hipMemcpyDeviceToHost);
	hipMemcpy(h_dy, d_dy, s*sizeof(t_def), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	// TimeVar t_3 = std::chrono::high_resolution_clock::now();
	
	// auto int_upload = std::chrono::duration_cast<std::chrono::milliseconds>( t_1 - t_0 ).count();
	auto int_exec = std::chrono::duration_cast<std::chrono::milliseconds>( t_1 - t_0 ).count();
	// auto int_download = std::chrono::duration_cast<std::chrono::milliseconds>( t_3 - t_2 ).count();
	// auto int_total = std::chrono::duration_cast<std::chrono::milliseconds>( t_3 - t_0 ).count();
	
	std::cout << "----" << s  << " : " << executions << "----" << std::endl;
	std::cout <<"exec:  " << int_exec << std::endl;
	free(h_a);
	free(h_b);
	free(h_Z);
	free(h_L);
	free(h_J);
	free(h_Y);
	free(h_dx);

	hipFree(d_a); 
	hipFree(d_b);
	hipFree(d_Z);
	hipFree(d_L);
	hipFree(d_J);
	hipFree(d_Y);
	hipFree(d_dx);
	hipFree(d_dz);
	hipFree(d_abs_dz);
	hipFree(d_dy);

	hipblasDestroy(cublas_handle);	
}
void devInfo()
{
	hipDeviceProp_t prop;
	int devcount;
	hipGetDeviceCount(&devcount);
	std::cout << "Devices found: " << devcount << std::endl;
	for(int i=0; i<devcount; i++)
	{
		hipGetDeviceProperties(&prop, i);
		std::cout << "------------------" << std::endl;
		std::cout << "Device: " << i << std::endl;
		std::cout << "------------------" << std::endl;
		std::cout << "Name:\t\t\t" << prop.name << std::endl;
		std::cout << "GlobalMemory:\t\t" << prop.totalGlobalMem << std::endl;
		std::cout << "WarpSize:\t\t" << prop.warpSize << std::endl;
		std::cout << "MaxThreadsPerBlock:\t" << prop.maxThreadsPerBlock << std::endl;
		std::cout << "MaxThreadsDim:\t\t" << prop.maxThreadsDim[0] << " : " << prop.maxThreadsDim[1] << " : " << prop.maxThreadsDim[2] << std::endl;
		std::cout << "MaxGridSize:\t\t" << prop.maxGridSize[0] << " : " << prop.maxGridSize[1] << " : " << prop.maxGridSize[2] << std::endl;
		std::cout << "MultiProcessorCount:\t" << prop.multiProcessorCount << std::endl;
	}	
}
long int getGlobalMemory()
{
	long int globalMemory = 0;
	hipDeviceProp_t prop;
	int devcount;
	hipGetDeviceCount(&devcount);
	if (devcount > 0)
	{
		hipGetDeviceProperties(&prop, 0);
		globalMemory = prop.totalGlobalMem;

	}
	return globalMemory;
}
long int calcRequiredMemory(int s)
{
	return (4*s*s + 6*s) * sizeof(t_def);
}

void single_execution_series()
{
	devInfo();
	long int globalMemory = getGlobalMemory();
	std::cout << globalMemory << std::endl;
	// SINGLE EXECUTIONS
	int size = 1000;
	int maxsize = 20000;
	while(true)
	{
		long int requiredMemory = calcRequiredMemory(size);
		if(requiredMemory > (long int) (globalMemory * 0.9) && size < maxsize)
		{
			break;
		}
		else
		{
			single_execution(size);
			std::cout << "Required Memory: " << requiredMemory * 1e-9 << std::endl;
			size+=1000;	
		}
	}
}
void multiple_executions_series(int times)
{
	devInfo();
	long int globalMemory = getGlobalMemory();
	std::cout << globalMemory << std::endl;
	int size = 1000;
	int maxsize = 20000;
	while(true)
	{
		long int requiredMemory = calcRequiredMemory(size);
		if(requiredMemory > (long int) (globalMemory * 0.9) && size < maxsize)
		{
			break;
		}
		else
		{
			multiple_executions(size, times);
			std::cout << "Required Memory: " << requiredMemory * 1e-9 << std::endl;
			size+=1000;	
		}
	}
}
int main()
{
	std::cout << "------------------------------------------------" << std::endl;
	std::cout << "Type: " << typeid(t_def).name() <<  std::endl;
	std::cout << "------------------------------------------------" << std::endl;
	single_execution_series();
	multiple_executions_series(100);

	return 0;
}
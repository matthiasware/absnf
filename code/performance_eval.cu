#include <hipblas.h>
#include <hipsolver.h>
#include <iostream>
#include "cuutils.h"
#include "absnf.h"
#include "utils.hpp"
#include <chrono>
#define t_def double

typedef std::chrono::high_resolution_clock::time_point TimeVar;
void test(int s)
{
	t_def *h_a = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_b = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_Z = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_L = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_J = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_Y = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_dx = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_dz = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_dy = (t_def *)malloc(s*sizeof(t_def));

	t_def *d_a; hipMalloc((void **)&d_a, s*sizeof(t_def));
	t_def *d_b; hipMalloc((void **)&d_b, s*sizeof(t_def));
	t_def *d_Z; hipMalloc((void **)&d_Z, s*s*sizeof(t_def));
	t_def *d_L; hipMalloc((void **)&d_L, s*s*sizeof(t_def));
	t_def *d_J; hipMalloc((void **)&d_J, s*s*sizeof(t_def));
	t_def *d_Y; hipMalloc((void **)&d_Y, s*s*sizeof(t_def));		
	t_def *d_dx; hipMalloc((void **)&d_dx, s*sizeof(t_def));
	t_def *d_dz; hipMalloc((void **)&d_dz, s*sizeof(t_def));
	t_def *d_abs_dz; hipMalloc((void **)&d_abs_dz, s*sizeof(t_def));
	t_def *d_dy; hipMalloc((void **)&d_dy, s*sizeof(t_def));

	utils::fillRandVector(h_a, s,-10,10);
	utils::fillRandVector(h_b, s,-10,10);
	utils::fillRandVector(h_Z, s*s,-10,10);
	utils::fillRandVector(h_J, s*s,-10,10);
	utils::fillRandVector(h_Y, s*s,-10,10);
	utils::fillRandVector(h_dx, s,-10,10);
	utils::fillRandMatrix(h_L, s,s,-10,10,0,utils::MATRIXOPT::LOWER);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	// TimeVar t_0 = std::chrono::high_resolution_clock::now();

	hipMemcpy(d_a, h_a,  s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b,  s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_Z, h_Z,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_L, h_L,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_J, h_J,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, h_Y,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_dx, h_dx, s*sizeof(t_def), hipMemcpyHostToDevice);

	int k = 1000;
	TimeVar t_0 = std::chrono::high_resolution_clock::now();
	for(int i=0; i<k; i++)
	{
		absnf::eval_core(cublas_handle, d_a, d_b,
			 		  	 d_Z, d_L,
			 		  	 d_J, d_Y,
			 		  	 d_dx,
			 		  	 s, s, s,
			 		  	 d_dz, d_dy,
			 		  	 d_abs_dz);
	}
	hipDeviceSynchronize();
	TimeVar t_1 = std::chrono::high_resolution_clock::now();
	
	hipMemcpy(h_dz, d_dz, s*sizeof(t_def), hipMemcpyDeviceToHost);
	hipMemcpy(h_dy, d_dy, s*sizeof(t_def), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	// TimeVar t_3 = std::chrono::high_resolution_clock::now();
	
	// auto int_upload = std::chrono::duration_cast<std::chrono::milliseconds>( t_1 - t_0 ).count();
	auto int_exec = std::chrono::duration_cast<std::chrono::milliseconds>( t_1 - t_0 ).count();
	// auto int_download = std::chrono::duration_cast<std::chrono::milliseconds>( t_3 - t_2 ).count();
	// auto int_total = std::chrono::duration_cast<std::chrono::milliseconds>( t_3 - t_0 ).count();
	
	std::cout << "----" << s  << " : " << k << "----" << std::endl;
	std::cout <<"exec:  " << int_exec << std::endl;
	free(h_a);
	free(h_b);
	free(h_Z);
	free(h_L);
	free(h_J);
	free(h_Y);
	free(h_dx);

	hipFree(d_a); 
	hipFree(d_b);
	hipFree(d_Z);
	hipFree(d_L);
	hipFree(d_J);
	hipFree(d_Y);
	hipFree(d_dx);
	hipFree(d_dz);
	hipFree(d_abs_dz);
	hipFree(d_dy);

	hipblasDestroy(cublas_handle);	
}

int main()
{
	for(int i=1000; i<=10000; i+=1000)
	{
		test(i);
	}

	return 0;
}
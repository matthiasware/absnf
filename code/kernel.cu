#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "device_launch_paraMeters.h"

#include<iostream>
#include<iomanip>
#include<stdlib.h>
#include<stdio.h>
#include<assert.h>

#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#include "Utilities.cuh"

#define BLOCK_SIZE 32

/***************/
/* COPY KERNEL */
/***************/
__global__ void copy_kernel(const double * __restrict d_in1, double * __restrict d_out1, const double * __restrict d_in2, double * __restrict d_out2, const int M, const int N) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i < N) && (j < N)) {
        d_out1[j * N + i] = d_in1[j * M + i];
        d_out2[j * N + i] = d_in2[j * M + i];
    }
}

/********/
/* MAIN */
/********/
int main(){

    // --- ASSUMPTION Nrows >= Ncols

    const int Nrows = 7;
    const int Ncols = 5;

    // --- cuSOLVE input/output parameters/arrays
    int work_size = 0;
    int *devInfo;           gpuErrchk(hipMalloc(&devInfo,          sizeof(int)));

    // --- CUDA solver initialization
    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);

    // --- CUBLAS initialization
    hipblasHandle_t cublas_handle;
    cublasSafeCall(hipblasCreate(&cublas_handle));

    // --- Setting the host, Nrows x Ncols matrix
    double *h_A = (double *)malloc(Nrows * Ncols * sizeof(double));
    for(int j = 0; j < Nrows; j++)
        for(int i = 0; i < Ncols; i++)
            h_A[j + i*Nrows] = (i + j*j) * sqrt((double)(i + j));

    // --- Setting the device matrix and moving the host matrix to the device
    double *d_A;            gpuErrchk(hipMalloc(&d_A,      Nrows * Ncols * sizeof(double)));
    gpuErrchk(hipMemcpy(d_A, h_A, Nrows * Ncols * sizeof(double), hipMemcpyHostToDevice));

    // --- CUDA QR initialization
    double *d_TAU;      gpuErrchk(hipMalloc((void**)&d_TAU, min(Nrows, Ncols) * sizeof(double)));
    cusolveSafeCall(hipsolverDnDgeqrf_bufferSize(solver_handle, Nrows, Ncols, d_A, Nrows, &work_size));
    double *work;   gpuErrchk(hipMalloc(&work, work_size * sizeof(double)));

    // --- CUDA GERF execution
    cusolveSafeCall(hipsolverDnDgeqrf(solver_handle, Nrows, Ncols, d_A, Nrows, d_TAU, work, work_size, devInfo));
    int devInfo_h = 0;  gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if (devInfo_h != 0) std::cout   << "Unsuccessful gerf execution\n\n";

    // --- At this point, the upper triangular part of A contains the elements of R. Showing this.
    gpuErrchk(hipMemcpy(h_A, d_A, Nrows * Ncols * sizeof(double), hipMemcpyDeviceToHost));
    for(int j = 0; j < Nrows; j++)
        for(int i = 0; i < Ncols; i++)
            if (i >= j) printf("R[%i, %i] = %f\n", j, i, h_A[j + i*Nrows]);

    // --- Initializing the output Q matrix (Of course, this step could be done by a kernel function directly on the device)
    double *h_Q = (double *)malloc(Nrows * Nrows * sizeof(double));
    for(int j = 0; j < Nrows; j++)
        for(int i = 0; i < Nrows; i++)
            if (j == i) h_Q[j + i*Nrows] = 1.;
            else        h_Q[j + i*Nrows] = 0.;

    double *d_Q;            gpuErrchk(hipMalloc(&d_Q,      Nrows * Nrows * sizeof(double)));
    gpuErrchk(hipMemcpy(d_Q, h_Q, Nrows * Nrows * sizeof(double), hipMemcpyHostToDevice));

    // --- CUDA QR execution
    cusolveSafeCall(hipsolverDnDormqr(solver_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_N, Nrows, Ncols, min(Nrows, Ncols), d_A, Nrows, d_TAU, d_Q, Nrows, work, work_size, devInfo));

    // --- At this point, d_Q contains the elements of Q. Showing this.
    gpuErrchk(hipMemcpy(h_Q, d_Q, Nrows * Nrows * sizeof(double), hipMemcpyDeviceToHost));
    printf("\n\n");
    for(int j = 0; j < Nrows; j++)
        for(int i = 0; i < Nrows; i++)
            printf("Q[%i, %i] = %f\n", j, i, h_Q[j + i*Nrows]);

    // --- Initializing the data matrix C (Of course, this step could be done by a kernel function directly on the device).
    // --- Notice that, in this case, only the first column of C contains actual data, the others being empty (zeroed). However, cuBLAS trsm
    //     has the capability of solving triangular linear systems with multiple right hand sides.
    double *h_C = (double *)calloc(Nrows * Nrows, sizeof(double));
    for(int j = 0; j < Nrows; j++)
        h_C[j] = 1.;

    double *d_C;            gpuErrchk(hipMalloc(&d_C,      Nrows * Nrows * sizeof(double)));
    gpuErrchk(hipMemcpy(d_C, h_C, Nrows * Nrows * sizeof(double), hipMemcpyHostToDevice));

    // --- CUDA QR execution
    cusolveSafeCall(hipsolverDnDormqr(solver_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, Nrows, Ncols, min(Nrows, Ncols), d_A, Nrows, d_TAU, d_C, Nrows, work, work_size, devInfo));

    // --- At this point, d_C contains the elements of Q^T * C, where C is the data vector. Showing this.
    // --- According to the above, only the first column of d_C makes sense.
    gpuErrchk(hipMemcpy(h_C, d_C, Nrows * Nrows * sizeof(double), hipMemcpyDeviceToHost));
    printf("\n\n");
    for(int j = 0; j < Nrows; j++)
        for(int i = 0; i < Nrows; i++)
            printf("C[%i, %i] = %f\n", j, i, h_C[j + i*Nrows]);

    // --- Reducing the linear system size
    double *d_R; gpuErrchk(hipMalloc(&d_R, Ncols * Ncols * sizeof(double)));
    double *h_B = (double *)malloc(Ncols * Ncols * sizeof(double));
    double *d_B; gpuErrchk(hipMalloc(&d_B, Ncols * Ncols * sizeof(double)));
    dim3 Grid(iDivUp(Ncols, BLOCK_SIZE), iDivUp(Ncols, BLOCK_SIZE));
    dim3 Block(BLOCK_SIZE, BLOCK_SIZE);
    copy_kernel<<<Grid, Block>>>(d_A, d_R, d_C, d_B, Nrows, Ncols);

    // --- Solving an upper triangular linear system
    const double alpha = 1.;
    cublasSafeCall(hipblasDtrsm(cublas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, Ncols, Ncols,
                               &alpha, d_R, Ncols, d_B, Ncols));

    gpuErrchk(hipMemcpy(h_B, d_B, Ncols * Ncols * sizeof(double), hipMemcpyDeviceToHost));

    printf("\n\n");
    for (int i=0; i<Ncols; i++) printf("B[%i] = %f\n", i, h_B[i]);

    hipsolverDnDestroy(solver_handle);

    return 0;

}

#include <hipblas.h>
#include <hipsolver.h>
#include "cuutils.h"
#include <vector>
#include "utils.hpp"
#define t_def double

int main()
{
    int m = 4;
    int n = 4;
    int s = 3;
    // m x n
    std::vector<t_def> h_A = {1, 1, 2 ,1,
                                 4, 1, 0, 1,
                                 3, 5, 1, 6,
                                 1, 1, 0, 1};
    // m x s                                 
    std::vector<t_def> h_B = {0, 0, 2,
                            4, 2, 0,
                            2, 1, 3,
                            0, 1, 3};

    // (m x s) = (m x n) x (m * s)
    // t_def *h_IJY =(t_def *)malloc(m*s*sizeof(t_def));
    t_def *d_A; hipMalloc((void **)&d_A, m*n*sizeof(t_def));
    t_def *d_B; hipMalloc((void **)&d_B, m*s*sizeof(t_def));
    t_def *d_TAU; hipMalloc((void **)&d_TAU, m*sizeof(t_def));
    // t_def *d_IJY; hipMalloc((void **)&d_IJY, m*s*sizeof(t_def));

    hipMemcpy(d_A, &h_A[0], m*n*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_B, &h_B[0], m*s*sizeof(t_def), hipMemcpyHostToDevice);
    // --------------------------------------------------------------
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);
    // --------------------------------------------------------------
    // CALCULATE WORKING SPACE 
    int lwork = 0;
    hipsolverDnDgeqrf_bufferSize(solver_handle,
                                m,n,
                                d_A, m,
                                &lwork);
    t_def *d_work; hipMalloc((void **)&d_work, sizeof(t_def)*lwork);
    int *devInfo; hipMalloc((void **)&devInfo, sizeof(int));
    
    // COMPUTE QR factorization
    hipsolverDnDgeqrf(solver_handle,
                     m,n,
                     d_A,
                     m,
                     d_TAU,
                     d_work,
                     lwork,
                     devInfo);
    // COMPUTE Q^T B
    hipsolverDnDormqr(solver_handle,
                     HIPBLAS_SIDE_LEFT,
                     HIPBLAS_OP_T,
                     m,
                     s,
                     m,
                     d_A,
                     m,
                     d_TAU,
                     d_B,
                     m,
                     d_work,
                     lwork,
                     devInfo);

    // --------------------------------------------------------------
    hipsolverDnDestroy(solver_handle);
    hipblasDestroy(cublas_handle);
    // --------------------------------------------------------------
    // hipMemcpy(h_IJY, d_IJY, m*s*sizeof(t_def), hipMemcpyDeviceToHost);
    // free(h_A);
    hipFree(d_A);
    hipFree(d_B);
    return 0;
}
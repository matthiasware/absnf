#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hipsolver.h>
#include "cuutils.h"
#include <iostream>
#include "absnf.h"
#include "utils.hpp"
#include <vector>
#define t_def double

// template <typename T>
// void modulus_core(hipblasHandle_t &cublas_handle,
//                       T *d_S, T *d_c, T *d_abs_dz, int s, T *d_dz)
// {
//     // d_dz = c
//     hipMemcpy(d_dz, d_c, s*sizeof(T), hipMemcpyDeviceToDevice);
//     // d_dz = beta * d_dz + S * d_abs_dz
//     double alpha = 1;
//     double beta = 1;
//     hipblasDgemv(cublas_handle,
//                 HIPBLAS_OP_N,
//                 s, s,
//                 &alpha,
//                 d_S, s,
//                 d_abs_dz, 1,
//                 &beta,
//                 d_dz, 1);
// }

// template <typename T>
// void modulus(hipblasHandle_t &cublas_handle, 
//                  T *d_S, T *d_c, T *d_dz,
//                  T *d_abs_dz, T *d_dz_old,
//                  int m, int s,
//                  int blocksize, int gridsize,
//                  int maxiter, double tol, bool verbose)
//     {
//         int i=0;
//         double diff = tol + 1;
//         while(i < maxiter && diff > tol)
//         {
//             // dz_old = dz
//             cuutils::check(hipMemcpy(d_dz_old, d_dz, s*sizeof(T), hipMemcpyDeviceToDevice));
//             // abs_dz = |dz|
//             cuutils::abs<<<gridsize,blocksize>>>(d_dz, d_abs_dz, s);
//             // dz = calculateDZ()
//             modulus_core(cublas_handle, d_S, d_c, d_abs_dz, s, d_dz);
//             // calculate diff
//             cuutils::vvSub<<<gridsize, blocksize>>>(d_dz, d_dz_old, d_dz_old, s);
//             cuutils::check(hipblasDnrm2(cublas_handle,
//                     s,
//                     d_dz_old,
//                     1,
//                     &diff));
//             if(verbose)
//                 std::cout << i << ": " << diff << std::endl;
//             i++;
//         }
// }
int main()
{
    int seed = 0;
    // int s = 5; // works seed=1
    int s = 200; // works
    t_def *h_a = (t_def *)malloc(s*sizeof(t_def));
    t_def *h_b = (t_def *)malloc(s*sizeof(t_def));
    t_def *h_Z = (t_def *)malloc(s*s*sizeof(t_def));
    t_def *h_L = (t_def *)malloc(s*s*sizeof(t_def));
    t_def *h_J = (t_def *)malloc(s*s*sizeof(t_def));
    t_def *h_Y = (t_def *)malloc(s*s*sizeof(t_def));
    t_def *h_dx = (t_def *)malloc(s*s*sizeof(t_def));
    t_def *h_dz = (t_def *)malloc(s*sizeof(t_def));
    t_def *h_dy = (t_def *)malloc(s*sizeof(t_def));

    t_def *d_a; hipMalloc((void **)&d_a, s*sizeof(t_def));
    t_def *d_b; hipMalloc((void **)&d_b, s*sizeof(t_def));
    t_def *d_Z; hipMalloc((void **)&d_Z, s*s*sizeof(t_def));
    t_def *d_L; hipMalloc((void **)&d_L, s*s*sizeof(t_def));
    t_def *d_J; hipMalloc((void **)&d_J, s*s*sizeof(t_def));
    t_def *d_Y; hipMalloc((void **)&d_Y, s*s*sizeof(t_def));       
    t_def *d_dx; hipMalloc((void **)&d_dx, s*sizeof(t_def));
    t_def *d_dz; hipMalloc((void **)&d_dz, s*sizeof(t_def));
    t_def *d_dz_solve; hipMalloc((void **)&d_dz_solve, s*sizeof(t_def));
    t_def *d_dz_old; hipMalloc((void **)&d_dz_old, s*sizeof(t_def));
    t_def *d_abs_dz; hipMalloc((void **)&d_abs_dz, s*sizeof(t_def));
    t_def *d_dy; hipMalloc((void **)&d_dy, s*sizeof(t_def));

    t_def *d_S; hipMalloc((void **)&d_S, s*s*sizeof(t_def));
    t_def *d_c; hipMalloc((void **)&d_c, s*sizeof(t_def));

    utils::fillRandVector(h_a, s,-1,1, seed, utils::VALUEOP::REAL);
    utils::fillRandVector(h_b, s, -1,1, seed, utils::VALUEOP::REAL);
    utils::fillRandVector(h_Z, s*s,-1,1,seed, utils::VALUEOP::REAL);
    utils::fillRandVector(h_Y, s*s,-1,1,seed, utils::VALUEOP::REAL);
    utils::fillRandVector(h_dx, s,-1,1,seed, utils::VALUEOP::REAL);
    utils::fillRandMatrix(h_J, s,s,-1,1,seed, utils::MATRIXOPT::INVERTIBLE, utils::VALUEOP::REAL);
    utils::fillVector(h_L, s*s,0.0);


    utils::printf_vector(h_a, s, "a");
    utils::printf_vector(h_b, s, "b");
    utils::printf_matrix(h_Z, s, s, "Z");
    utils::printf_matrix(h_L, s, s, "L");
    utils::printf_matrix(h_J, s, s, "J");
    utils::printf_matrix(h_Y, s, s, "Y");
    utils::printf_vector(h_dx, s, "dx");

    utils::rowColConversion(h_Z, s, s, true);
    utils::rowColConversion(h_Y, s, s, true);
    utils::rowColConversion(h_J, s, s, true);

    hipMemcpy(d_a, h_a,  s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b,  s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_Z, h_Z,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_L, h_L,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_J, h_J,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, h_Y,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_dx, h_dx, s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_dz_solve, h_a, s*sizeof(t_def), hipMemcpyHostToDevice);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);

    int gridsize, blocksize;
    cuutils::getGridBlockSize(&gridsize, &blocksize);

    // EVALUATE ABSNF
    absnf::eval_core(cublas_handle, d_a, d_b,
                         d_Z, d_L,
                         d_J, d_Y,
                         d_dx,
                         s, s, s,
                         d_dz, d_dy,
                         d_abs_dz);

    cuutils::printf_vector(d_dz, s, "d_dz_eval");
    // SOLVE ABSNF
    // ADJUST b
    cuutils::vvSub<<<gridsize, blocksize>>>(d_b, d_dy, d_b, s);
    cuutils::printf_vector(d_b, s, "d_b");
    
    absnf::calculate_S_and_c(cublas_handle, solver_handle,
                      d_a, d_b, d_Z, d_L, d_J, d_Y, 
                      s, s,
                      d_c, d_S);

    cuutils::printf_vector(d_S, s*s, "d_S");
    cuutils::printf_vector(d_c, s, "d_c");

    absnf::modulus(cublas_handle, d_S, d_c, d_dz_solve,
            d_abs_dz, d_dz_old, s, s,blocksize, gridsize,
            10000, 1e-4, true);

    cuutils::printf_vector(d_dz, s, "d_dz");
    cuutils::printf_vector(d_dz_solve, s, "d_dz_solve");

    free(h_a);
    free(h_b);
    free(h_Z);
    free(h_L);
    free(h_J);
    free(h_Y);
    free(h_dx);

    hipFree(d_a); 
    hipFree(d_b);
    hipFree(d_Z);
    hipFree(d_L);
    hipFree(d_S);
    hipFree(d_c);
    hipFree(d_J);
    hipFree(d_Y);
    hipFree(d_dx);
    hipFree(d_dz);
    hipFree(d_dz_solve);
    hipFree(d_dz_old);
    hipFree(d_abs_dz);
    hipFree(d_dy);

    hipblasDestroy(cublas_handle);   

	return 0;
}
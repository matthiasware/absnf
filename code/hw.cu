#include "hip/hip_runtime.h"
#include <iostream>

// every function that runs on the device and is called from the host
// is a kernel
// a divive function runs at and is called from the device
__global__ void add(int *a, int *b, int *c)
{
	*c = *a + *b;
}

int main(void)
{
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	// allocate memory
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	# setup data
	a = 2;
	b = 7;

	// copy data
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	// Launch Kernel
	add <<<1, 1>>>(d_a, d_b, d_c);

	// Copy result back
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	// clean
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
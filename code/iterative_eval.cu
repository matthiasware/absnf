#include <hipblas.h>
#include "utils.hpp"

#define t_def double

int main()
{
	int n=4, s=3;

	t_def *h_a = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_Z = (t_def *)malloc(s*n*sizeof(t_def));
	t_def *h_L = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_dx =(t_def *)malloc(n*sizeof(t_def));
	t_def *h_dz =(t_def *)malloc(s*sizeof(t_def));

	// HOST MEMORY
	utils::fillVector(h_dz, s, (t_def) 0);
	utils::fillRandVector(h_a, s, 0, 5, 1, utils::VALUEOP::INT);
	utils::fillRandVector(h_dx, n, 0, 5, 2, utils::VALUEOP::INT);
	utils::fillRandMatrix(h_Z, s, n, 0, 5, 3, utils::MATRIXOPT::NONE, utils::VALUEOP::INT);
	utils::fillRandMatrix(h_L, s, s, 1, 5, 4, utils::MATRIXOPT::LOWER, utils::VALUEOP::INT);

	// DEVICE MEMORY
	t_def *d_a; hipMalloc((void **)&d_a, s*sizeof(t_def));
	t_def *d_Z; hipMalloc((void **)&d_Z, s*n*sizeof(t_def));
	t_def *d_L; hipMalloc((void **)&d_L, s*s*sizeof(t_def));
	t_def *d_dx; hipMalloc((void **)&d_dx, n*sizeof(t_def));
	t_def *d_dz; hipMalloc((void **)&d_dz, s*sizeof(t_def));
	t_def *d_abs_dz; hipMalloc((void **)&d_dz, s*sizeof(t_def));

	//COPY DATA
	hipMemcpy(d_a, h_a,  s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_Z, h_Z,  s*n*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_L, h_L,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_dx, h_dx, n*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_dz, h_dz, s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_abs_dz, h_dz, s*sizeof(t_def), hipMemcpyHostToDevice);


	utils::printf_vector(h_a, s);
	utils::printf_vector(h_dz, s);
	utils::printf_vector(h_dx, n);
	utils::printf_matrix(h_Z, s, n);
	utils::printf_matrix(h_L, s, s);


	// FREE STUFF
	free(h_a); free(h_L); free(h_Z);
	free(h_dz); free(h_dx);
	hipFree(d_a);
	hipFree(d_Z);
	hipFree(d_L);
	hipFree(d_dx);
	hipFree(d_dz);
	return 0;
}
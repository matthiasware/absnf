#include <hipblas.h>
#include <hipsolver.h>
#include <iostream>
#include "cuutils.h"
#include "absnf.h"
#include "utils.hpp"
#include <chrono>
#define t_def double

typedef std::chrono::high_resolution_clock::time_point TimeVar;
void test(int s)
{
	t_def *h_a = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_b = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_Z = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_L = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_J = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_Y = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_dz = (t_def *)malloc(s*sizeof(t_def));

	t_def *d_a; hipMalloc((void **)&d_a, s*sizeof(t_def));
	t_def *d_b; hipMalloc((void **)&d_b, s*sizeof(t_def));
	t_def *d_Z; hipMalloc((void **)&d_Z, s*s*sizeof(t_def));
	t_def *d_L; hipMalloc((void **)&d_L, s*s*sizeof(t_def));
	t_def *d_J; hipMalloc((void **)&d_J, s*s*sizeof(t_def));
	t_def *d_Y; hipMalloc((void **)&d_Y, s*s*sizeof(t_def));		
	t_def *d_dz; hipMalloc((void **)&d_dz, s*sizeof(t_def));
	t_def *d_gamma; hipMalloc((void **)&d_gamma, s*sizeof(t_def));
	t_def *d_Gamma; hipMalloc((void **)&d_Gamma, s*s*sizeof(t_def));
	t_def *d_Tss; hipMalloc((void **)&d_Tss, s*s*sizeof(t_def));
	t_def *d_I; hipMalloc((void **)&d_I, s*s*sizeof(t_def));
	t_def *d_K; hipMalloc((void **)&d_K, s*s*sizeof(t_def));

	utils::fillRandVector(h_a, s,-10,10);
	utils::fillRandVector(h_b, s,-10,10);
	utils::fillRandVector(h_Z, s*s,-10,10);
	utils::fillRandMatrix(h_L, s,s,-10,10,0,utils::MATRIXOPT::LOWER);
	utils::fillRandVector(h_J, s*s,-10,10);
	utils::fillRandVector(h_Y, s*s,-10,10);
	utils::fillRandVector(h_dz, s,-10,10);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	TimeVar t_0 = std::chrono::high_resolution_clock::now();

	hipMemcpy(d_a, h_a,  s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b,  s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_Z, h_Z,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_L, h_L,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_J, h_J,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, h_Y,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_dz, h_dz, s*sizeof(t_def), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	TimeVar t_1 = std::chrono::high_resolution_clock::now();
	auto time_copy = std::chrono::duration_cast<std::chrono::milliseconds>( t_1 - t_0 ).count();
	int gridsize, blocksize;
	cuutils::getGridBlockSize(&gridsize, &blocksize);

	t_0 = std::chrono::high_resolution_clock::now();
	absnf::gradient_core(cublas_handle,
							 d_a, d_b, d_Z, d_L, d_J,
							 d_Y, d_dz, d_Tss, d_I, d_K,
							 s, s, s,
							 gridsize, blocksize,
							 d_gamma, d_Gamma);
	
	hipDeviceSynchronize();
	t_1 = std::chrono::high_resolution_clock::now();
	auto time_exec = std::chrono::duration_cast<std::chrono::milliseconds>( t_1 - t_0 ).count();
	
	// TimeVar t_3 = std::chrono::high_resolution_clock::now();
	
	// auto int_upload = std::chrono::duration_cast<std::chrono::milliseconds>( t_1 - t_0 ).count();
	// auto int_download = std::chrono::duration_cast<std::chrono::milliseconds>( t_3 - t_2 ).count();
	// auto int_total = std::chrono::duration_cast<std::chrono::milliseconds>( t_3 - t_0 ).count();
	
	std::cout << "---------------" << std::endl;
	std::cout << "s=m=n: " << s  << std::endl;
	std::cout << "mem:   " << time_copy << std::endl;
	std::cout << "exec:  " << time_exec << std::endl;
	free(h_a);
	free(h_b);
	free(h_Z);
	free(h_L);
	free(h_J);
	free(h_Y);
	free(h_dz);

	hipFree(d_a); 
	hipFree(d_b);
	hipFree(d_Z);
	hipFree(d_L);
	hipFree(d_J);
	hipFree(d_Y);
	hipFree(d_dz);
	hipFree(d_Tss);
	hipFree(d_I);
	hipFree(d_K);
	hipFree(d_gamma);
	hipFree(d_Gamma);

	hipblasDestroy(cublas_handle);
}
void test_1000(int s)
{
	t_def *h_a = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_b = (t_def *)malloc(s*sizeof(t_def));
	t_def *h_Z = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_L = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_J = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_Y = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *h_dz = (t_def *)malloc(s*sizeof(t_def));

	t_def *d_a; hipMalloc((void **)&d_a, s*sizeof(t_def));
	t_def *d_b; hipMalloc((void **)&d_b, s*sizeof(t_def));
	t_def *d_Z; hipMalloc((void **)&d_Z, s*s*sizeof(t_def));
	t_def *d_L; hipMalloc((void **)&d_L, s*s*sizeof(t_def));
	t_def *d_J; hipMalloc((void **)&d_J, s*s*sizeof(t_def));
	t_def *d_Y; hipMalloc((void **)&d_Y, s*s*sizeof(t_def));		
	t_def *d_dz; hipMalloc((void **)&d_dz, s*sizeof(t_def));
	t_def *d_gamma; hipMalloc((void **)&d_gamma, s*sizeof(t_def));
	t_def *d_Gamma; hipMalloc((void **)&d_Gamma, s*s*sizeof(t_def));
	t_def *d_Tss; hipMalloc((void **)&d_Tss, s*s*sizeof(t_def));
	t_def *d_I; hipMalloc((void **)&d_I, s*s*sizeof(t_def));
	t_def *d_K; hipMalloc((void **)&d_K, s*s*sizeof(t_def));

	utils::fillRandVector(h_a, s,-10,10);
	utils::fillRandVector(h_b, s,-10,10);
	utils::fillRandVector(h_Z, s*s,-10,10);
	utils::fillRandMatrix(h_L, s,s,-10,10,0,utils::MATRIXOPT::LOWER);
	utils::fillRandVector(h_J, s*s,-10,10);
	utils::fillRandVector(h_Y, s*s,-10,10);
	utils::fillRandVector(h_dz, s,-10,10);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);


	hipMemcpy(d_a, h_a,  s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b,  s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_Z, h_Z,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_L, h_L,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_J, h_J,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, h_Y,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_dz, h_dz, s*sizeof(t_def), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	int gridsize, blocksize;
	cuutils::getGridBlockSize(&gridsize, &blocksize);

	TimeVar t_0 = std::chrono::high_resolution_clock::now();
	for(int i=0; i<100; i++)
	{
		absnf::gradient_core(cublas_handle,
							 d_a, d_b, d_Z, d_L, d_J,
							 d_Y, d_dz, d_Tss, d_I, d_K,
							 s, s, s,
							 gridsize, blocksize,
							 d_gamma, d_Gamma);
	}
	hipDeviceSynchronize();
	TimeVar t_1 = std::chrono::high_resolution_clock::now();
	auto time_exec = std::chrono::duration_cast<std::chrono::milliseconds>( t_1 - t_0 ).count();
	
	// TimeVar t_3 = std::chrono::high_resolution_clock::now();
	
	// auto int_upload = std::chrono::duration_cast<std::chrono::milliseconds>( t_1 - t_0 ).count();
	// auto int_download = std::chrono::duration_cast<std::chrono::milliseconds>( t_3 - t_2 ).count();
	// auto int_total = std::chrono::duration_cast<std::chrono::milliseconds>( t_3 - t_0 ).count();
	
	std::cout << "----1000--------" << std::endl;
	std::cout << "s=m=n: " << s  << std::endl;
	std::cout << "exec:  " << time_exec << std::endl;
	free(h_a);
	free(h_b);
	free(h_Z);
	free(h_L);
	free(h_J);
	free(h_Y);
	free(h_dz);

	hipFree(d_a); 
	hipFree(d_b);
	hipFree(d_Z);
	hipFree(d_L);
	hipFree(d_J);
	hipFree(d_Y);
	hipFree(d_dz);
	hipFree(d_Tss);
	hipFree(d_I);
	hipFree(d_K);
	hipFree(d_gamma);
	hipFree(d_Gamma);

	hipblasDestroy(cublas_handle);
}
void test_gridsize(int s)
{

}
int main()
{
	test(5000);
	return 0;
}
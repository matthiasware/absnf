#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hipsolver.h>
#include "cuutils.h"
#include "absnf.h"
#include "utils.hpp"
#include <vector>
#define t_def double

void test()
{
    int m = 4;
    int n = 4;
    int s = 3;
    // m x n
    std::vector<t_def> h_A = {1, 1, 2 ,1,
                                 4, 1, 0, 1,
                                 3, 5, 1, 6,
                                 1, 1, 0, 1};
    utils::rowColConversion(&h_A[0], m, n, true);                                
    // m x s                                 
    std::vector<t_def> h_B = {0, 0, 2,
                            4, 2, 0,
                            2, 1, 3,
                            0, 1, 3};
    utils::rowColConversion(&h_B[0], m, s, true);

    // (m x s) = (m x n) x (m * s)
    // t_def *h_IJY =(t_def *)malloc(m*s*sizeof(t_def));
    t_def *d_A; hipMalloc((void **)&d_A, m*n*sizeof(t_def));
    t_def *d_B; hipMalloc((void **)&d_B, m*s*sizeof(t_def));
    t_def *d_TAU; hipMalloc((void **)&d_TAU, m*sizeof(t_def));
    // t_def *d_IJY; hipMalloc((void **)&d_IJY, m*s*sizeof(t_def));

    hipMemcpy(d_A, &h_A[0], m*n*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_B, &h_B[0], m*s*sizeof(t_def), hipMemcpyHostToDevice);
    // --------------------------------------------------------------
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);
    // --------------------------------------------------------------
    // CALCULATE WORKING SPACE 
    int lwork = 0;
    hipsolverDnDgeqrf_bufferSize(solver_handle,
                                m,n,
                                d_A, m,
                                &lwork);
    t_def *d_work; hipMalloc((void **)&d_work, sizeof(t_def)*lwork);
    int *devInfo; hipMalloc((void **)&devInfo, sizeof(int));
    
    // COMPUTE QR factorization
    hipsolverDnDgeqrf(solver_handle,
                     m,n,
                     d_A,
                     m,
                     d_TAU,
                     d_work,
                     lwork,
                     devInfo);
    // COMPUTE Q^T B
    hipsolverDnDormqr(solver_handle,
                     HIPBLAS_SIDE_LEFT,
                     HIPBLAS_OP_T,
                     m,
                     s,
                     m,
                     d_A,
                     m,
                     d_TAU,
                     d_B,
                     m,
                     d_work,
                     lwork,
                     devInfo);
    // COMPUTE X = R \ Q^T * B
    double one = 1;
    hipblasDtrsm(cublas_handle,
                HIPBLAS_SIDE_LEFT,
                HIPBLAS_FILL_MODE_UPPER,
                HIPBLAS_OP_N,
                HIPBLAS_DIAG_NON_UNIT,
                m,
                s,
                &one,
                d_A,
                m,
                d_B,
                m);
    cuutils::printf_vector(d_B, m*s, "X");
    // --------------------------------------------------------------
    hipsolverDnDestroy(solver_handle);
    hipblasDestroy(cublas_handle);
    // --------------------------------------------------------------
    // hipMemcpy(h_IJY, d_IJY, m*s*sizeof(t_def), hipMemcpyDeviceToHost);
    // free(h_A);
    hipFree(d_A);
    hipFree(d_B);
}
void test2()
{
    int m = 4;
    int n = 4;
    int s = 3;
    // m x n
    std::vector<t_def> h_A = {1, 1, 2 ,1,
                                 4, 1, 0, 1,
                                 3, 5, 1, 6,
                                 1, 1, 0, 1};
    utils::rowColConversion(&h_A[0], m, n, true);                                
    // m x s                                 
    std::vector<t_def> h_B = {0, 0, 2,
                            4, 2, 0,
                            2, 1, 3,
                            0, 1, 3};
    utils::rowColConversion(&h_B[0], m, s, true);

    // m
    std::vector<t_def> h_b = {-275, -126, -484, -450};
    // (m x s) = (m x n) x (m * s)
    t_def *d_A; hipMalloc((void **)&d_A, m*n*sizeof(t_def));
    t_def *d_B; hipMalloc((void **)&d_B, m*s*sizeof(t_def));
    t_def *d_b; hipMalloc((void **)&d_b, m*sizeof(t_def));

    hipMemcpy(d_A, &h_A[0], m*n*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_B, &h_B[0], m*s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b[0], m*sizeof(t_def), hipMemcpyHostToDevice);
    // --------------------------------------------------------------
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);
    // --------------------------------------------------------------
    // solveAXeqB(cublas_handle, solver_handle, d_A, d_B, m, s);
    // cuutils::printf_vector(d_B, m*s, "X");
    // solveAXeqB(cublas_handle, solver_handle, d_A, d_b, m, 1);
    // cuutils::printf_vector(d_b, m, "x");
    // --------------------------------------------------------------
    hipsolverDnDestroy(solver_handle);
    hipblasDestroy(cublas_handle);
    // --------------------------------------------------------------
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_b);
}

int main()
{
    int m = 4;
    int s = 3;
    // m x m
    std::vector<t_def> h_J = {100, 1, 2 ,1,
                              4, 120, 0, 1,
                              3, 5, 120, 6,
                              1, 1, 0, 130};
    utils::rowColConversion(&h_J[0], m, m, true);                                
    // m x s                                 
    std::vector<t_def> h_Y = {0, 0, 2,
                              4, 2, 0,
                              2, 1, 3,
                              0, 1, 3};
    utils::rowColConversion(&h_Y[0], m, s, true);

    // s x s
    std::vector<t_def> h_L = {0,0,0,
                            1,0,0,
                            0,1,0};
    utils::rowColConversion(&h_L[0], s, s, true);

    // s x m
    std::vector<t_def> h_Z = {-4,  0, -4 , 1,
                               3,  0, -2, -3,
                              -3, -4, -4,  0};
    utils::rowColConversion(&h_Z[0], s, m, true);

    // m
    // std::vector<t_def> h_b = {-275, -126, -484, -450};
    // b = b - y IMPORTANT
    std::vector<t_def> h_b = {-223, -432, -200, -48};

    // s
    std::vector<t_def> h_a = {4, 4, -3};

    // dz_start
    // t_def *h_dz = (t_def *) malloc(s * sizeof(t_def));
    // utils::fillRandVector(h_dz, s, -10, 10, 4);
    t_def h_dz[] = {-1.59449432,  9.28890523,  9.39411967};

    // (m x s) = (m x n) x (m * s)
    t_def *d_J; hipMalloc((void **)&d_J, m*m*sizeof(t_def));
    t_def *d_Y; hipMalloc((void **)&d_Y, m*s*sizeof(t_def));
    t_def *d_Z; hipMalloc((void **)&d_Z, s*m*sizeof(t_def));
    t_def *d_b; hipMalloc((void **)&d_b, m*sizeof(t_def));
    t_def *d_a; hipMalloc((void **)&d_a, s*sizeof(t_def));
    t_def *d_L; hipMalloc((void **)&d_L, s*s*sizeof(t_def));
    t_def *d_S; hipMalloc((void **)&d_S, s*s*sizeof(t_def));
    t_def *d_c; hipMalloc((void **)&d_c, s*sizeof(t_def));
    t_def *d_dz; hipMalloc((void **)&d_dz, s*sizeof(t_def));
    t_def *d_abs_dz; hipMalloc((void **)&d_abs_dz, s*sizeof(t_def));
    t_def *d_dz_old; hipMalloc((void **)&d_dz_old, s*sizeof(t_def));
    t_def *d_dx; hipMalloc((void **)&d_dx, m*sizeof(t_def));

    hipMemcpy(d_J, &h_J[0], m*m*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, &h_Y[0], m*s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_Z, &h_Z[0], s*m*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b[0], m*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_a, &h_a[0], s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_L, &h_L[0], s*s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_dz, h_dz, s*sizeof(t_def), hipMemcpyHostToDevice);

    // --------------------------------------------------------------
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);
    // Calculate S, c
    // calculate_S(cublas_handle, solver_handle,
    //             d_L, d_Z, d_J, d_Y, m, s, d_S);
    
    // hipMemcpy(d_J, &h_J[0], m*m*sizeof(t_def), hipMemcpyHostToDevice);
    // hipMemcpy(d_Y, &h_Y[0], m*s*sizeof(t_def), hipMemcpyHostToDevice);
    // hipMemcpy(d_Z, &h_Z[0], s*m*sizeof(t_def), hipMemcpyHostToDevice);
    // hipMemcpy(d_b, &h_b[0], m*sizeof(t_def), hipMemcpyHostToDevice);

    // calculate_c(cublas_handle, solver_handle,
    //             d_a, d_Z, d_J, d_b, m, s, d_c);

    absnf::calculate_S_and_c(cublas_handle, solver_handle,
                      d_a, d_b, d_Z, d_L, d_J, d_Y, 
                      m, s,
                      d_c, d_S); 

    cuutils::printf_vector(d_S, s*s, "d_S");
    cuutils::printf_vector(d_c, s, "d_c");
    //--------------------------------------------------------------
    cuutils::printf_vector(d_dz, s, "dz_start");

    int gridsize, blocksize;
    cuutils::getGridBlockSize(&gridsize, &blocksize);
    std::cout << gridsize << " : " << blocksize << std::endl;
    // int maxiter = 100;
    // int i = 0;
    // double tol = 1e-8;
    // double diff = tol + 1;
    // // cuutils::printf_vector(d_dz, s, "d_dz");
    // while(i < maxiter && diff > tol)
    // {
    //     // dz_old = dz
    //     hipMemcpy(d_dz_old, d_dz, s*sizeof(t_def), hipMemcpyDeviceToDevice);
    //     // abs_dz = |dz|
    //     cuutils::abs<<<gridsize,blocksize>>>(d_dz, d_abs_dz, s);
    //     std::cout << "----" << i << "----"  << std::endl;
    //     cuutils::printf_vector(d_dz_old, s, "d_dz_old");
    //     cuutils::printf_vector(d_abs_dz, s, "d_abs_dz");
    //     // dz = calculateDZ()
    //     modulus(cublas_handle, d_S, d_c, d_abs_dz, s, d_dz);
    //     cuutils::printf_vector(d_dz, s, "d_dz_new");
    //     // calculate diff
    //     cuutils::vvSub<<<gridsize, blocksize>>>(d_dz, d_dz_old, d_dz_old, s);
    //     hipblasDnrm2(cublas_handle,
    //                 s,
    //                 d_dz_old,
    //                 1,
    //                 &diff);
    //     // std::cout << i << ": " << diff << std::endl;
    //     i++;
    // }
    absnf::modulus(cublas_handle, d_S, d_c, d_dz,
                   d_abs_dz, d_dz_old, m, s,blocksize, gridsize);
    cuutils::printf_vector(d_dz, s, "Result: d_dz"); 
    // --------------------------------------------------------------
    hipsolverDnDestroy(solver_handle);
    hipblasDestroy(cublas_handle);
    // --------------------------------------------------------------
    hipFree(d_J);
    hipFree(d_Y);
    hipFree(d_Z);
    hipFree(d_b);
    hipFree(d_a);
    hipFree(d_L);
    hipFree(d_S);
    hipFree(d_c);
    hipFree(d_dx);
    hipFree(d_dz);
    hipFree(d_abs_dz);
    hipFree(d_dz_old);
	return 0;
}
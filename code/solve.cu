#include <hipblas.h>
#include <hipsolver.h>
#include "cuutils.h"
#include <vector>
#include "utils.hpp"
#define t_def double

void test()
{
    int m = 4;
    int n = 4;
    int s = 3;
    // m x n
    std::vector<t_def> h_A = {1, 1, 2 ,1,
                                 4, 1, 0, 1,
                                 3, 5, 1, 6,
                                 1, 1, 0, 1};
    utils::rowColConversion(&h_A[0], m, n, true);                                
    // m x s                                 
    std::vector<t_def> h_B = {0, 0, 2,
                            4, 2, 0,
                            2, 1, 3,
                            0, 1, 3};
    utils::rowColConversion(&h_B[0], m, s, true);

    // (m x s) = (m x n) x (m * s)
    // t_def *h_IJY =(t_def *)malloc(m*s*sizeof(t_def));
    t_def *d_A; hipMalloc((void **)&d_A, m*n*sizeof(t_def));
    t_def *d_B; hipMalloc((void **)&d_B, m*s*sizeof(t_def));
    t_def *d_TAU; hipMalloc((void **)&d_TAU, m*sizeof(t_def));
    // t_def *d_IJY; hipMalloc((void **)&d_IJY, m*s*sizeof(t_def));

    hipMemcpy(d_A, &h_A[0], m*n*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_B, &h_B[0], m*s*sizeof(t_def), hipMemcpyHostToDevice);
    // --------------------------------------------------------------
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);
    // --------------------------------------------------------------
    // CALCULATE WORKING SPACE 
    int lwork = 0;
    hipsolverDnDgeqrf_bufferSize(solver_handle,
                                m,n,
                                d_A, m,
                                &lwork);
    t_def *d_work; hipMalloc((void **)&d_work, sizeof(t_def)*lwork);
    int *devInfo; hipMalloc((void **)&devInfo, sizeof(int));
    
    // COMPUTE QR factorization
    hipsolverDnDgeqrf(solver_handle,
                     m,n,
                     d_A,
                     m,
                     d_TAU,
                     d_work,
                     lwork,
                     devInfo);
    // COMPUTE Q^T B
    hipsolverDnDormqr(solver_handle,
                     HIPBLAS_SIDE_LEFT,
                     HIPBLAS_OP_T,
                     m,
                     s,
                     m,
                     d_A,
                     m,
                     d_TAU,
                     d_B,
                     m,
                     d_work,
                     lwork,
                     devInfo);
    // COMPUTE X = R \ Q^T * B
    double one = 1;
    hipblasDtrsm(cublas_handle,
                HIPBLAS_SIDE_LEFT,
                HIPBLAS_FILL_MODE_UPPER,
                HIPBLAS_OP_N,
                HIPBLAS_DIAG_NON_UNIT,
                m,
                s,
                &one,
                d_A,
                m,
                d_B,
                m);
    cuutils::printf_vector(d_B, m*s, "X");
    // --------------------------------------------------------------
    hipsolverDnDestroy(solver_handle);
    hipblasDestroy(cublas_handle);
    // --------------------------------------------------------------
    // hipMemcpy(h_IJY, d_IJY, m*s*sizeof(t_def), hipMemcpyDeviceToHost);
    // free(h_A);
    hipFree(d_A);
    hipFree(d_B);
}
void test2()
{
    int m = 4;
    int n = 4;
    int s = 3;
    // m x n
    std::vector<t_def> h_A = {1, 1, 2 ,1,
                                 4, 1, 0, 1,
                                 3, 5, 1, 6,
                                 1, 1, 0, 1};
    utils::rowColConversion(&h_A[0], m, n, true);                                
    // m x s                                 
    std::vector<t_def> h_B = {0, 0, 2,
                            4, 2, 0,
                            2, 1, 3,
                            0, 1, 3};
    utils::rowColConversion(&h_B[0], m, s, true);

    // m
    std::vector<t_def> h_b = {-275, -126, -484, -450};
    // (m x s) = (m x n) x (m * s)
    t_def *d_A; hipMalloc((void **)&d_A, m*n*sizeof(t_def));
    t_def *d_B; hipMalloc((void **)&d_B, m*s*sizeof(t_def));
    t_def *d_b; hipMalloc((void **)&d_b, m*sizeof(t_def));

    hipMemcpy(d_A, &h_A[0], m*n*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_B, &h_B[0], m*s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b[0], m*sizeof(t_def), hipMemcpyHostToDevice);
    // --------------------------------------------------------------
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);
    // --------------------------------------------------------------
    // solveAXeqB(cublas_handle, solver_handle, d_A, d_B, m, s);
    // cuutils::printf_vector(d_B, m*s, "X");
    // solveAXeqB(cublas_handle, solver_handle, d_A, d_b, m, 1);
    // cuutils::printf_vector(d_b, m, "x");
    // --------------------------------------------------------------
    hipsolverDnDestroy(solver_handle);
    hipblasDestroy(cublas_handle);
    // --------------------------------------------------------------
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_b);
}

	/** Solves AX=B with QR Decomposition
	
		INPUT:
		@param d_A: device mem (m*m) column major
		@param d_B: device mem (m*s) column major
		OUTPUT:
		@param d_B: X
        @param d_A: QR decomposition
	*/
template <typename T>
void solveAXeqB(hipblasHandle_t &cublas_handle,
			    hipsolverHandle_t &solver_handle,
			   T *d_A, T *d_B, int m, int s)
{
    int *d_devInfo; hipMalloc((void **)&d_devInfo, sizeof(int));
    // scaling factors for householder reflectors
    T *d_TAU; hipMalloc((void **)&d_TAU, m*sizeof(T));

    // calculate working space
    int h_swork = 0; // working space
    hipsolverDnDgeqrf_bufferSize(solver_handle,
                                m,m,
                                d_A, m,
                                &h_swork);
    T *d_work; hipMalloc((void **)&d_work, sizeof(T)*h_swork);

    // ----------------------------------
    // d_A <-- A=Q*R  d_TAU <--
    // ----------------------------------
    hipsolverDnDgeqrf(solver_handle,
                     m,m,
                     d_A,
                     m,
                     d_TAU,
                     d_work,
                     h_swork,
                     d_devInfo);
    // ----------------------------------
    // d_B <- Q^T B
    // ----------------------------------
    hipsolverDnDormqr(solver_handle,
                     HIPBLAS_SIDE_LEFT,
                     HIPBLAS_OP_T,
                     m,
                     s,
                     m,
                     d_A,
                     m,
                     d_TAU,
                     d_B,
                     m,
                     d_work,
                     h_swork,
                     d_devInfo);
    // ----------------------------------
    // d_B <- X = R \ Q^T * B
    // ----------------------------------
    double alpha = 1;
    hipblasDtrsm(cublas_handle,
                HIPBLAS_SIDE_LEFT,
                HIPBLAS_FILL_MODE_UPPER,
                HIPBLAS_OP_N,
                HIPBLAS_DIAG_NON_UNIT,
                m,
                s,
                &alpha,
                d_A,
                m,
                d_B,
                m);

    hipFree(d_devInfo);
    hipFree(d_TAU);
}
template <typename T>
void calculate_S(hipblasHandle_t &cublas_handle,
                hipsolverHandle_t &solver_handle,
                T *d_L, T *d_Z, T *d_J, T *d_Y, int m, int s, T *d_S)
{
    // cuutils::printf_vector(d_Y, m*s, "Y");
    // S = L
    hipMemcpy(d_S, d_L, s*s*sizeof(T), hipMemcpyDeviceToDevice);
    // d_Y <- J^{-1}*Y
    solveAXeqB(cublas_handle, solver_handle, d_J, d_Y, m, s);
    // S = S + (-1) Z*d_Y
    double alpha = -1;
    double beta = 1;
    hipblasDgemm(cublas_handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                s,s,m,
                &alpha,
                d_Z,
                s,
                d_Y,
                m,
                &beta,
                d_S,
                s);
}
template <typename T>
void calculate_c(hipblasHandle_t &cublas_handle,
                 hipsolverHandle_t &solver_handle,
                 T *d_a, T *d_Z, T *d_J, T *d_b, int m, int s, T *d_c)
{

    // c = a
    hipMemcpy(d_c, d_a, s*sizeof(T), hipMemcpyDeviceToDevice);

    // d_b <- J^{-1}*b
    solveAXeqB(cublas_handle, solver_handle, d_J, d_b, m, 1);

    double alpha = -1;
    double beta = 1;
    hipblasDgemv(cublas_handle,
                HIPBLAS_OP_N,
                s, m,
                &alpha,
                d_Z,
                s,
                d_b,
                1,
                &beta,
                d_c,
                1);
}

int main()
{
    int m = 4;
    int s = 3;
    // m x m
    std::vector<t_def> h_J = {1, 1, 2 ,1,
                              4, 1, 0, 1,
                              3, 5, 1, 6,
                              1, 1, 0, 1};
    utils::rowColConversion(&h_J[0], m, m, true);                                
    // m x s                                 
    std::vector<t_def> h_Y = {0, 0, 2,
                              4, 2, 0,
                              2, 1, 3,
                              0, 1, 3};
    utils::rowColConversion(&h_Y[0], m, s, true);

    // s x s
    std::vector<t_def> h_L = {0,0,0,
                            1,0,0,
                            0,1,0};
    utils::rowColConversion(&h_L[0], s, s, true);

    // s x m
    std::vector<t_def> h_Z = {-4,  0, -4 , 1,
                               3,  0, -2, -3,
                              -3, -4, -4,  0};
    utils::rowColConversion(&h_Z[0], s, m, true);

    // m
    std::vector<t_def> h_b = {-275, -126, -484, -450};

    // s
    std::vector<t_def> h_a = {4, 4, -3};

    // (m x s) = (m x n) x (m * s)
    t_def *d_J; hipMalloc((void **)&d_J, m*m*sizeof(t_def));
    t_def *d_Y; hipMalloc((void **)&d_Y, m*s*sizeof(t_def));
    t_def *d_Z; hipMalloc((void **)&d_Z, s*m*sizeof(t_def));
    t_def *d_b; hipMalloc((void **)&d_b, m*sizeof(t_def));
    t_def *d_a; hipMalloc((void **)&d_a, s*sizeof(t_def));
    t_def *d_L; hipMalloc((void **)&d_L, s*s*sizeof(t_def));
    t_def *d_S; hipMalloc((void **)&d_S, s*s*sizeof(t_def));
    t_def *d_c; hipMalloc((void **)&d_c, s*sizeof(t_def));

    hipMemcpy(d_J, &h_J[0], m*m*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, &h_Y[0], m*s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_Z, &h_Z[0], s*m*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b[0], m*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_a, &h_a[0], s*sizeof(t_def), hipMemcpyHostToDevice);
    hipMemcpy(d_L, &h_L[0], s*s*sizeof(t_def), hipMemcpyHostToDevice);

    // --------------------------------------------------------------
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);
    // --------------------------------------------------------------
    // calculate_S(cublas_handle, solver_handle,
    //             d_L, d_Z, d_J, d_Y, m, s, d_S);
    calculate_c(cublas_handle, solver_handle,
                d_a, d_Z, d_J, d_b, m, s, d_c);
    // cuutils::printf_vector(d_S, s*s, "S");
    // --------------------------------------------------------------
    hipsolverDnDestroy(solver_handle);
    hipblasDestroy(cublas_handle);
    // --------------------------------------------------------------
    hipFree(d_J);
    hipFree(d_Y);
    hipFree(d_Z);
    hipFree(d_b);
    hipFree(d_a);
    hipFree(d_L);
    hipFree(d_S);
    hipFree(d_c);
	return 0;
}
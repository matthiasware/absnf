#include <hipblas.h>
#include "absnf.h"
#include <vector>
#include "utils.hpp"
#define t_def double

bool test_eval_Singular(t_def *h_a, t_def *h_b, 
			   			t_def *h_Z, t_def *h_L, 
			   			t_def *h_J, t_def *h_Y,
			   			t_def *h_dx,
			   			int m, int n, int s,
			   			t_def *h_dz_expected, t_def *h_dy_expected)
{
	// convert to column major storage
	utils::rowColConversion(h_Z, s, n, true);
	utils::rowColConversion(h_J, m, n, true);
	utils::rowColConversion(h_Y, m, s, true);

	// results
	t_def *h_dz =(t_def *)malloc(s*sizeof(t_def));
	t_def *h_dy = (t_def *)malloc(m*sizeof(t_def));

	// calculate results
	absnf::eval(h_a, h_b,
			    h_Z, h_L, 
			    h_J, h_Y, 
			    h_dx, 
			    m, n, s, 
			    h_dz, h_dy);

	// compare results
	if(!utils::vectors_equals(h_dz, h_dz_expected, s, false))
		return false;
	if(!utils::vectors_equals(h_dy, h_dy_expected, m, false))
		return false;
	return true;
}
bool test_initTss_Singular(t_def *h_L, 
			   			   t_def *h_dz,
						   int s,
			   			   t_def *h_Tss_expected)
{
	bool success = true;
	t_def *h_Tss = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *d_L; hipMalloc((void **)&d_L, s*s*sizeof(t_def));
	t_def *d_dz; hipMalloc((void **)&d_dz, s*sizeof(t_def));
	t_def *d_Tss; hipMalloc((void **)&d_Tss, s*s*sizeof(t_def));

	hipMemcpy(d_L, h_L,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
	hipMemcpy(d_dz, h_dz, s*sizeof(t_def), hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	int gridsize, blocksize;
	cuutils::getGridBlockSize(&gridsize, &blocksize);
	absnf::initTss <<<gridsize, blocksize >>>(d_Tss,d_L, d_dz, s, s*s);
	hipMemcpy(h_Tss, d_Tss, s*s*sizeof(t_def), hipMemcpyDeviceToHost);

	if(!utils::vectors_equals(h_Tss, h_Tss_expected, s*s, false))
		success = false;


	hipFree(d_L);
	hipFree(d_dz);
	hipFree(d_Tss);
	free(h_Tss);
	return success;
}
bool test_getTriangularInverse_Singular(t_def *h_A, t_def *h_I_expected, int s)
{
	bool success = true;
	t_def *h_I_actual = (t_def *)malloc(s*s*sizeof(t_def));
	t_def *d_A; hipMalloc((void **)&d_A, s*s*sizeof(t_def));
	t_def *d_I; hipMalloc((void **)&d_I, s*s*sizeof(t_def));

	hipMemcpy(d_A, h_A, s*s*sizeof(t_def), hipMemcpyHostToDevice);
	// ..............
	int gridsize, blocksize;
	cuutils::getGridBlockSize(&gridsize, &blocksize);
	absnf::initIdentity<<<gridsize, blocksize>>>(d_I, s);
	// ..............
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	absnf::getTriangularInverse(handle, d_A, d_I, s);
	// ..............
	hipMemcpy(h_I_actual, d_I, s*s*sizeof(t_def), hipMemcpyDeviceToHost);
	if(!utils::vectors_equals(h_I_actual, h_I_expected, s*s))
		success = false;
	// utils::printf_vector(h_I_actual,s*s, "Inverse");
	hipFree(d_A);
	hipFree(d_I);
	free(h_I_actual);
	return success;
}
bool test_getTriangularInverse()
{
	bool success = true;
	int s=4;
	std::vector<t_def> h_matrix = {1, 0, 0, 0,
								   4, 1, 0, 0,
								   8, 0, 1, 0,
								   2, 0, -7, 1};
	std::vector<t_def> h_I_expected = { 1, 0, 0, 0,
								       -4, 1, 0, 0,
								       -8, 0, 1, 0,
								       -58,0, 7, 1};
    // utils::rowColConversion(&h_matrix[0], s, s, true);
	test_getTriangularInverse_Singular(&h_matrix[0], &h_I_expected[0], s);

	return success;
}
bool test_initTss()
{
	int s=4;
	std::vector<t_def> L = {0, 0, 0, 0,
						    4, 0, 0, 0,
						    8, 9, 0, 0,
						    2, 1, 7, 0};
	std::vector<t_def> dz = {-1, 0, 1, -1};
	std::vector<t_def> Tss_expected = {1, 0, 0, 0,
									   4, 1, 0, 0,
									   8, 0, 1, 0,
									   2, 0, -7, 1};
	test_initTss_Singular(&L[0], &dz[0], s, &Tss_expected[0]);
	return true;
}
bool test_initIdentity()
{
	bool success = true;
	int s=4;
	std::vector<t_def> h_I_expected = {1,0,0,0,
							 0,1,0,0,
							 0,0,1,0,
							 0,0,0,1};
	t_def *h_I_actual = (t_def *) malloc(s*s*sizeof(t_def));
	t_def *d_I; hipMalloc((void **)&d_I, s*s*sizeof(t_def));

	//  ----------------------------------
	int gridsize, blocksize;
	cuutils::getGridBlockSize(&gridsize, &blocksize);
	absnf::initIdentity<<<gridsize,blocksize>>>(d_I, s);
	//  ----------------------------------

	hipMemcpy(h_I_actual, d_I, s*s*sizeof(t_def), hipMemcpyDeviceToHost);
	if(!utils::vectors_equals(h_I_actual, &h_I_expected[0], s*s, false))
		success = false;

	hipFree(d_I);
	free(h_I_actual);
	return success;

}
bool test_eval()
{
	int n=4;
	int s=3;
	int m=2;
	// n
	std::vector<t_def> a = {4, 4,-3};
	// m
	std::vector<t_def> b = {4, 4};
	// s * n
	std::vector<t_def> Z = {-4,  0, -4,  1,
							 3,  0, -2, -3,
							-3, -4, -4,  0};
	// s * s
	std::vector<t_def> L = {0, 0, 0,
						    4, 0, 0,
						    0, 4, 0};
	// m * n
	std::vector<t_def> J = {0, 0, 2, 0,
						    4, 2, 0, 1};
	// m * s
	std::vector<t_def> Y = {0, 0, 2,
							4, 2, 0};
	// n
	std::vector<t_def> dx = {-3, 4, 4, 0};
	// s
	std::vector<t_def> dz_expected = {0, -13, 26};
	// m
	std::vector<t_def> dy_expected = {64, 26};

	if(!test_eval_Singular(&a[0], &b[0], &Z[0], &L[0], &J[0], &Y[0], 
			               &dx[0], m, n, s, &dz_expected[0], &dy_expected[0]))
		return false;


	// n = 5;
	// s = 4;
	// m = 3;
	// a = {0, 4, -3, 10};
	// b = {-8, 11, 7};
	// Z = {-4,  0, -4,  1 ,-1,
	// 	  3,  0, -2, -3, -21,
	// 	 -3, -4, -4, -1,  33,
	// 	 -9,  0, -5,  3,  4};
	// L = {0, 0, 0, 0,
	//      4, 0, 0, 0,
	//      8, 9, 0, 0,
	//      2, 1, 7, 0};
	// J = {0, 0, 2, 1, 3,
	//      4, 2, 0, 1, 2,
	//      1, 3, -2, 1, 8};
	// Y = {0, 0, 2, 1,
	// 	 4, 2, 0, 4,
	// 	 1, 4, 7, 4};


	return true;
}


int main()
{	
	test_eval();
	test_initTss();
	test_initIdentity();
	test_getTriangularInverse();
	return 0;
}
// bool test_initTss_Singular(t_def *h_a, t_def *h_b, 
// 			   			   t_def *h_Z, t_def *h_L, 
// 			   			   t_def *h_J, t_def *h_Y,
// 			   			   t_def *h_dz,
// 			   			   int m, int n, int s,
// 			   			   t_def *h_Tss_expected)
// {
// 	t_def *h_Tss = (t_def *)malloc(s*s*sizeof(t_def));
// 	t_def *d_a; hipMalloc((void **)&d_a, s*sizeof(t_def));
// 	t_def *d_b; hipMalloc((void **)&d_b, m*sizeof(t_def));
// 	t_def *d_Z; hipMalloc((void **)&d_Z, s*n*sizeof(t_def));
// 	t_def *d_L; hipMalloc((void **)&d_L, s*s*sizeof(t_def));
// 	t_def *d_J; hipMalloc((void **)&d_J, m*n*sizeof(t_def));
// 	t_def *d_Y; hipMalloc((void **)&d_Y, m*s*sizeof(t_def));		
// 	t_def *d_dz; hipMalloc((void **)&d_dz, s*sizeof(t_def));
// 	t_def *d_Tss; hipMalloc((void **)&d_Tss, s*s*sizeof(t_def));

// 	hipMemcpy(d_a, h_a,  s*sizeof(t_def), hipMemcpyHostToDevice);
// 	hipMemcpy(d_b, h_b,  m*sizeof(t_def), hipMemcpyHostToDevice);
// 	hipMemcpy(d_Z, h_Z,  s*n*sizeof(t_def), hipMemcpyHostToDevice);
// 	hipMemcpy(d_L, h_L,  s*s*sizeof(t_def), hipMemcpyHostToDevice);
// 	hipMemcpy(d_J, h_J,  m*n*sizeof(t_def), hipMemcpyHostToDevice);
// 	hipMemcpy(d_Y, h_Y,  m*s*sizeof(t_def), hipMemcpyHostToDevice);
// 	hipMemcpy(d_dz, h_dz, s*sizeof(t_def), hipMemcpyHostToDevice);

// 	hipblasHandle_t handle;
// 	hipblasCreate(&handle);
// 	//  ----------------------------------
// 	int gridsize, blocksize;
// 	cuutils::getGridBlockSize(&gridsize, &blocksize);
// 	absnf::initTss <<<gridsize, blocksize >>>(d_Tss,d_L, d_dz, s, s*s);
// 	hipMemcpy(h_Tss, d_Tss, s*s*sizeof(t_def), hipMemcpyDeviceToHost);

// 	// ----------------------------------

// 	hipFree(d_a); 
// 	hipFree(d_b);
// 	hipFree(d_Z);
// 	hipFree(d_L);
// 	hipFree(d_J);
// 	hipFree(d_Y);
// 	hipFree(d_dz);
// 	hipFree(d_Tss);
// 	free(h_Tss);
// 	return true;
// }